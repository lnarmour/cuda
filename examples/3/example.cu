#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

/*

We can invoke add() on the device in parallel
Each parallel invocation is called a "block"
* the set of blocks in called a "grid"
* each invocation can refer to its block index using "blockIdx.x"

*/

__global__ void add(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

#define N 512
#define MAX_INT 500

int random_ints(int* array, int N) {
	for (int i=0; i<N; i++) {
		array[i] = rand() % MAX_INT + 1;
	}
	return 0;
}

int main(void) {
	int *a, *b, *c;             // host copies of a, b, c
	int *d_a, *d_b, *d_c;    // device copies
	int size = sizeof(int) * N;

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// set up input valies
	a = (int *)malloc(size);
	b = (int *)malloc(size);
	c = (int *)malloc(size);
	random_ints(a, N);
	random_ints(b, N);

	// copy inputs to device
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	// launch add() kernel on CPU
	add<<<1,1>>>(d_a, d_b, d_c);

	// copy result back to host
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	for (int i=0; i<N; i++) {
		if (i == 11) {
			printf("...\n");
		}
		if (10 < i < N-1) {			
			continue;
		}
		printf("a[%d] = %d, b[%d] = %d, c[%d] = %d\n", i, a[i], i, b[i], i, c[i]);

	}

	// cleanup
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}

